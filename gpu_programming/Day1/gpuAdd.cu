#include <stdio.h>
#include <hip/hip_runtime.h>
// CUDA Kernel
__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

/**
 * Host main routine
 */
int
main(void)
{
    int numElements = 15;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    float a[numElements],b[numElements],c[numElements];
    float *a_gpu,*b_gpu,*c_gpu;

    hipMalloc((void **)&a_gpu, size);
    hipMalloc((void **)&b_gpu, size);
    hipMalloc((void **)&c_gpu, size);

    for (int i=0;i<numElements;++i ){
    
    	a[i] = i*i;
    	b[i] = i;
    
    }
    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    hipMemcpy(a_gpu, a, size, hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b, size, hipMemcpyHostToDevice);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(a_gpu, b_gpu, c_gpu, numElements);

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    hipMemcpy(c, c_gpu, size, hipMemcpyDeviceToHost);

    // Free device global memory
    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(c_gpu);
    
    for (int i=0;i<numElements;++i ){
    
    	printf("%f \n",c[i]);
    
    }
    printf("Done\n");
    return 0;
}

